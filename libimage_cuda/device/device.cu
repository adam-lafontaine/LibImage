#include "device.hpp"
#include "cuda_def.cuh"

#include <cassert>
#include <cstddef>

#ifdef CUDA_PRINT_ERROR

#include <cstdio>
#include <cstring>

#endif


static void check_error(hipError_t err, cstr label = "")
{
    if (err == hipSuccess)
    {
        return;
    }

    #ifdef CUDA_PRINT_ERROR
    #ifndef	NDEBUG

    printf("\n*** CUDA ERROR ***\n\n");
    printf("%s", hipGetErrorString(err));

    if (std::strlen(label))
    {
        printf("\n%s", label);
    }
    
    printf("\n\n******************\n\n");

    #endif
    #endif
}


namespace cuda
{
    bool device_malloc(ByteBuffer& buffer, size_t n_bytes)
    {
        assert(n_bytes);
        assert(!buffer.data);

        if (!n_bytes || buffer.data)
        {
            return false;
        }

        hipError_t err = hipMalloc((void**)&(buffer.data), n_bytes);
        check_error(err, "malloc");

        bool result = err == hipSuccess;

        if (result)
        {
            buffer.capacity = n_bytes;
            buffer.size = 0;
        }
        
        return result;
    }


    bool unified_malloc(ByteBuffer& buffer, size_t n_bytes)
    {
        assert(n_bytes);
        assert(!buffer.data);

        if (!n_bytes || buffer.data)
        {
            return false;
        }

        hipError_t err = hipMallocManaged((void**)&(buffer.data), n_bytes);
        check_error(err, "unified_malloc");

        bool result = err == hipSuccess;

        if (result)
        {
            buffer.capacity = n_bytes;
            buffer.size = 0;
        }
        
        return result;
    }


    bool host_malloc(ByteBuffer& buffer, size_t n_bytes)
    {
        assert(n_bytes);
        assert(!buffer.data);

        if (!n_bytes || buffer.data)
        {
            return false;
        }

        buffer.data = (u8*)std::malloc(n_bytes);

        if (!buffer.data)
        {
            return false;
        }

        return true;
    }


    bool free(ByteBuffer& buffer)
    {
        buffer.capacity = 0;
        buffer.size = 0;

        if (buffer.data)
        {
            hipError_t err = hipFree(buffer.data);
            check_error(err, "free");

            buffer.data = nullptr;

            return err == hipSuccess;
        }

        return true;
    }


    bool host_free(ByteBuffer& buffer)
    {
        buffer.capacity = 0;
        buffer.size = 0;

        if (buffer.data)
        {
            std::free(buffer.data);
        }

        return true;
    }


    u8* push_bytes(ByteBuffer& buffer, size_t n_bytes)
    {
        assert(buffer.data);
        assert(buffer.capacity);
        assert(buffer.size < buffer.capacity);

        auto is_valid = 
            buffer.data &&
            buffer.capacity &&
            buffer.size < buffer.capacity;

        auto bytes_available = (buffer.capacity - buffer.size) >= n_bytes;
        assert(bytes_available);

        if (!is_valid || !bytes_available)
        {
            return nullptr;
        }

        auto data = buffer.data + buffer.size;

        buffer.size += n_bytes;

        return data;
    }


    bool pop_bytes(ByteBuffer& buffer, size_t n_bytes)
    {
        assert(buffer.data);
        assert(buffer.capacity);
        assert(buffer.size <= buffer.capacity);
        assert(n_bytes <= buffer.capacity);
        assert(n_bytes <= buffer.size);

        auto is_valid = 
            buffer.data &&
            buffer.capacity &&
            buffer.size <= buffer.capacity &&
            n_bytes <= buffer.capacity &&
            n_bytes <= buffer.size;

        if (is_valid)
        {
            buffer.size -= n_bytes;
            return true;
        }

        return false;
    }
    

    bool memcpy_to_device(const void* host_src, void* device_dst, size_t n_bytes)
    {
        hipError_t err = hipMemcpy(device_dst, host_src, n_bytes, hipMemcpyHostToDevice);
        check_error(err, "memcpy_to_device");

        return err == hipSuccess;
    }


    bool memcpy_to_host(const void* device_src, void* host_dst, size_t n_bytes)
    {
        hipError_t err = hipMemcpy(host_dst, device_src, n_bytes, hipMemcpyDeviceToHost);
        check_error(err, "memcpy_to_host");

        return err == hipSuccess;
    }


    bool no_errors(cstr label)
    {
        #ifndef	NDEBUG

        hipError_t err = hipGetLastError();
        check_error(err, label);

        return err == hipSuccess;

        #else

        return true;

        #endif
    }


    bool launch_success(cstr label)
    {
        #ifndef	NDEBUG

        hipError_t err = hipDeviceSynchronize();
        check_error(err, label);

        return err == hipSuccess;

        #else

        return true;

        #endif
    }
}