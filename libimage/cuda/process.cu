#include "hip/hip_runtime.h"
#include "cuda_def.cuh"
#include "process.hpp"
#include "convolve.cuh"

#include <cassert>

constexpr int THREADS_PER_BLOCK = 1024;


class Point2Du32
{
public:
	u32 x;
	u32 y;
};


class Point2Dr32
{
public:
	r32 x;
	r32 y;
};


namespace libimage
{
    
#ifndef LIBIMAGE_NO_COLOR

namespace gpu
{
/*******************************/


GPU_FUNCTION
static u8 blend_linear(u8 s, u8 c, r32 a)
{
    
    auto const sf = (r32)(s);
    auto const cf = (r32)(c);

    auto blended = a * cf + (1.0f - a) * sf;

    return (u8)(blended);
}


GPU_FUNCTION
static pixel_t to_pixel(u8 r, u8 g, u8 b)
{
    pixel_t p{};

    p.red = r;
    p.green = g;
    p.blue = b;
    p.alpha = 255;

    return p;
}


GPU_FUNCTION
static pixel_t get_color(device_image_t const& image, r32 x, r32 y)
{
    auto zero = 0.0f;
    auto width = (r32)image.width;
    auto height = (r32)image.height;

    if (x < zero || x >= width || y < zero || y >= height)
    {
        return gpu::to_pixel(0, 0, 0);
    }

    //auto row_begin = image.data; // + (size_t)((u32)floorf(y) * image.width);

    int ux = __float2int_rd(2.5f);

    //return row_begin[(u32)floorf(x)];

    return gpu::to_pixel(0, 0, 0);
}


__device__
static int get_int()
{
    int ux = __float2int_rd(2.5f);

    return 0;
}


GPU_FUNCTION
static Point2Dr32 find_rotation_src(Point2Du32 const& pt, Point2Du32 const& origin, r32 theta_rotate)
{
	auto dx_dst = (r32)pt.x - (r32)origin.x;
	auto dy_dst = (r32)pt.y - (r32)origin.y;

	auto radius = hypotf(dx_dst, dy_dst);

	auto theta_dst = atan2f(dy_dst, dx_dst);
	auto theta_src = theta_dst - theta_rotate;

	auto dx_src = radius * cosf(theta_src);
	auto dy_src = radius * sinf(theta_src);

	Point2Dr32 pt_src{};
	pt_src.x = (r32)origin.x + dx_src;
	pt_src.y = (r32)origin.y + dy_src;

	return pt_src;
}



/*******************************/
}




GPU_KERNAL
static void gpu_alpha_blend_linear(pixel_t* src, pixel_t* current, pixel_t* dst, u32 n_elements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= n_elements)
    {
        return;
    }

    auto const a = (r32)(src[i].alpha) / 255.0f;
    auto red = gpu::blend_linear(src[i].red, current[i].red, a);
    auto green = gpu::blend_linear(src[i].green, current[i].green, a);
    auto blue = gpu::blend_linear(src[i].blue, current[i].blue, a);

    dst[i] = { red, green, blue, 255 };
}


GPU_KERNAL
static void gpu_rotate(device_image_t src, device_image_t dst, Point2Du32 const& origin, r32 theta_rotate, u32 n_elements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= n_elements)
    {
        return;
    }

    auto dst_pixel_offset = (u32)i;

    assert(dst_pixel_offset < dst.width * dst.height);

    Point2Du32 pt{};
    pt.y = dst_pixel_offset / dst.width;
    pt.x = dst_pixel_offset - pt.y * dst.width;

    auto pt_src = gpu::find_rotation_src(pt, origin, theta_rotate);

    i = gpu::get_int();

    dst.data[dst_pixel_offset] = gpu::get_color(src, pt_src.x, pt_src.y);
}






#endif // !LIBIMAGE_NO_COLOR

#ifndef LIBIMAGE_NO_GRAYSCALE

GPU_CONSTEXPR_FUNCTION r32 div16(int i) { return i / 16.0f; }

GPU_GLOBAL_CONSTANT r32 GAUSS_3X3[]
{
    div16(1), div16(2), div16(1),
    div16(2), div16(4), div16(2),
    div16(1), div16(2), div16(1),
};


GPU_CONSTEXPR_FUNCTION r32 div256(int i) { return i / 256.0f; }

GPU_GLOBAL_CONSTANT r32 GAUSS_5X5[]
{
    div256(1), div256(4),  div256(6),  div256(4),  div256(1),
    div256(4), div256(16), div256(24), div256(16), div256(4),
    div256(6), div256(24), div256(36), div256(24), div256(6),
    div256(4), div256(16), div256(24), div256(16), div256(4),
    div256(1), div256(4),  div256(6),  div256(4),  div256(1),
};


GPU_GLOBAL_CONSTANT r32 GRAD_X_3X3[]
{
    1.0f, 0.0f, -1.0f,
    2.0f, 0.0f, -2.0f,
    1.0f, 0.0f, -1.0f,
};


GPU_GLOBAL_CONSTANT r32 GRAD_Y_3X3[]
{
    1.0f,  2.0f,  1.0f,
    0.0f,  0.0f,  0.0f,
    -1.0f, -2.0f, -1.0f,
};


GPU_KERNAL
static void gpu_binarize(u8* src, u8* dst, u8 threshold, u32 n_elements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= n_elements)
    {
        return;
    }

    dst[i] = src[i] >= threshold ? 255 : 0;
}


GPU_KERNAL
static void gpu_blur(u8* src, u8* dst, u32 width, u32 height)
{
    u32 n_elements = width * height;
    u32 i = u32(blockDim.x * blockIdx.x + threadIdx.x);
    if (i >= n_elements)
    {
        return;
    }

    auto g3x3 = (r32*)GAUSS_3X3;
    auto g5x5 = (r32*)GAUSS_5X5;

    if(is_outer_edge(width, height, i))
    {
        dst[i] = src[i];
    }
    else if(is_inner_edge(width, height, i))
    {
        dst[i] = convolve_3x3(src, width, height, i, g3x3);
    }
    else
    {
        dst[i] = convolve_5x5(src, width, height, i, g5x5);
    }
}


GPU_KERNAL
static void gpu_edges(u8* src, u8* dst, u32 width, u32 height, u8 threshold)
{
    u32 n_elements = width * height;
    u32 i = u32(blockDim.x * blockIdx.x + threadIdx.x);
    if (i >= n_elements)
    {
        return;
    }

    if(is_outer_edge(width, height, i))
    {
        dst[i] = 0;
    }
    else
    {
        auto gx = convolve_3x3(src, width, height, i, GRAD_X_3X3);
        auto gy = convolve_3x3(src, width, height, i, GRAD_Y_3X3);
        auto g = (u8)(std::hypot(gx, gy));
        dst[i] = g < threshold ? 0 : 255;
    }
}


GPU_KERNAL
static void gpu_gradients(u8* src, u8* dst, u32 width, u32 height)
{
    u32 n_elements = width * height;
    u32 i = u32(blockDim.x * blockIdx.x + threadIdx.x);
    if (i >= n_elements)
    {
        return;
    }

    if(is_outer_edge(width, height, i))
    {
        dst[i] = 0;
    }
    else
    {
        auto gx = convolve_3x3(src, width, height, i, GRAD_X_3X3);
        auto gy = convolve_3x3(src, width, height, i, GRAD_Y_3X3);
        dst[i] = (u8)(std::hypot(gx, gy));
    }
}


GPU_FUNCTION
static u8 lerp_clamp(u8 src_low, u8 src_high, u8 dst_low, u8 dst_high, u8 val)
{
    if (val < src_low)
    {
        return dst_low;
    }
    else if (val > src_high)
    {
        return dst_high;
    }

    auto const ratio = ((r64)(val) - src_low) / (src_high - src_low);

    assert(ratio >= 0.0);
    assert(ratio <= 1.0);

    auto const diff = ratio * (dst_high - dst_low);

    return dst_low + (u8)(diff);
}


GPU_KERNAL
static void gpu_transform_contrast(u8* src, u8* dst, u8 src_low, u8 src_high, u32 n_elements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= n_elements)
    {
        return;
    }

    u8 dst_low = 0;
	u8 dst_high = 255;

    dst[i] = lerp_clamp(src_low, src_high, dst_low, dst_high, src[i]);
}

#endif // !LIBIMAGE_NO_GRAYSCALE

#ifndef LIBIMAGE_NO_COLOR
#ifndef LIBIMAGE_NO_GRAYSCALE

GPU_FUNCTION
static u8 rgb_grayscale_standard(u8 red, u8 green, u8 blue)
{
    return (u8)(0.299f * red + 0.587f * green + 0.114f * blue);
}


GPU_KERNAL
static void gpu_transform_grayscale(pixel_t* src, u8* dst, u32 n_elements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= n_elements)
    {
        return;
    }

    u8 r = src[i].red;
    u8 g = src[i].green;
    u8 b = src[i].blue;

    dst[i] = rgb_grayscale_standard(r, g, b);
}

#endif // !LIBIMAGE_NO_GRAYSCALE
#endif // !LIBIMAGE_NO_COLOR	

}


namespace libimage
{

#ifndef LIBIMAGE_NO_COLOR

    bool alpha_blend(device_image_t const& src, device_image_t const& current, device_image_t const& dst)
    {
        assert(src.data);
        assert(src.width);
        assert(src.height);
        assert(current.data);
        assert(current.width == src.width);
        assert(current.height == src.height);
        assert(dst.data);
        assert(dst.width == src.width);
        assert(dst.height == src.height);

        u32 n_elements = src.width * src.height;
        int threads_per_block = THREADS_PER_BLOCK;
        int blocks = (n_elements + threads_per_block - 1) / threads_per_block;

        bool proc;

        proc = cuda_no_errors();
        assert(proc);

        gpu_alpha_blend_linear<<<blocks, threads_per_block>>>(
            src.data, 
            current.data, 
            dst.data, 
            n_elements);

        proc &= cuda_launch_success();
        assert(proc);

        return proc;
    }


    bool alpha_blend(device_image_t const& src, device_image_t const& current_dst)
    {
        assert(src.data);
        assert(src.width);
        assert(src.height);
        assert(current_dst.data);
        assert(current_dst.width == src.width);
        assert(current_dst.height == src.height);

        u32 n_elements = src.width * src.height;
        int threads_per_block = THREADS_PER_BLOCK;
        int blocks = (n_elements + threads_per_block - 1) / threads_per_block;

        bool proc;

        proc = cuda_no_errors();
        assert(proc);

        gpu_alpha_blend_linear<<<blocks, threads_per_block>>>(
            src.data, 
            current_dst.data, 
            current_dst.data, 
            n_elements);

        proc &= cuda_launch_success();
        assert(proc);

        return proc;
    }


    bool rotate(device_image_t const& src, device_image_t const& dst, u32 origin_x, u32 origin_y, r32 theta)
    {
        Point2Du32 origin = { origin_x, origin_y };
        u32 n_elements = dst.width * dst.height;
        int threads_per_block = THREADS_PER_BLOCK;
        int blocks = (n_elements + threads_per_block - 1) / threads_per_block;

        bool proc;

        proc = cuda_no_errors();
        assert(proc);

        gpu_rotate<<<blocks, threads_per_block>>>(src, dst, origin, theta, n_elements);

        proc &= cuda_launch_success();
        assert(proc);

        return proc;
    }


#endif // !LIBIMAGE_NO_COLOR	

#ifndef LIBIMAGE_NO_GRAYSCALE

    bool binarize(gray::device_image_t const& src, gray::device_image_t const& dst, u8 min_threshold)
    {
        assert(src.data);
        assert(src.width);
        assert(src.height);
        assert(dst.data);
        assert(dst.width == src.width);
        assert(dst.height == src.height);

        u32 n_elements = src.width * src.height;
        int threads_per_block = THREADS_PER_BLOCK;
        int blocks = (n_elements + threads_per_block - 1) / threads_per_block;

        bool proc;

        proc = cuda_no_errors();
        assert(proc);

        gpu_binarize<<<blocks, threads_per_block>>>(
            src.data, 
            dst.data,
            min_threshold, 
            n_elements);

        proc &= cuda_launch_success();
        assert(proc);

        return proc;
    }


    bool blur(gray::device_image_t const& src, gray::device_image_t const& dst)
    {
        assert(src.data);
        assert(src.width);
        assert(src.height);
        assert(dst.data);
        assert(dst.width == src.width);
        assert(dst.height == src.height);

        u32 n_elements = src.width * src.height;
        int threads_per_block = THREADS_PER_BLOCK;
        int blocks = (n_elements + threads_per_block - 1) / threads_per_block;

        bool proc;

        proc = cuda_no_errors();
        assert(proc);

        gpu_blur<<<blocks, threads_per_block>>>(
            src.data, 
            dst.data, 
            src.width, 
            src.height);
        
        proc &= cuda_launch_success();
        assert(proc);

        return proc;
    }


    bool edges(gray::device_image_t const& src, gray::device_image_t const& dst, u8 threshold, gray::device_image_t const& temp)
    {
        assert(src.data);
        assert(src.width);
        assert(src.height);
        assert(dst.data);
        assert(dst.width == src.width);
        assert(dst.height == src.height);
        assert(temp.data);
        assert(temp.width == src.width);
        assert(temp.height == src.height);

        u32 n_elements = src.width * src.height;
        int threads_per_block = THREADS_PER_BLOCK;
        int blocks = (n_elements + threads_per_block - 1) / threads_per_block;

        bool proc;

        proc = cuda_no_errors();
        assert(proc);

        gpu_blur<<<blocks, threads_per_block>>>(
            src.data, 
            temp.data,
            src.width, 
            src.height);
        
        proc &= cuda_launch_success();
        assert(proc);

        proc &= cuda_no_errors();
        assert(proc);

        gpu_edges<<<blocks, threads_per_block>>>(
            temp.data,
            dst.data,
            src.width,
            src.height,
            threshold);

        proc &= cuda_launch_success();
        assert(proc);

        return proc;
    }


    bool edges(gray::device_image_t const& src, gray::device_image_t const& dst, u8 threshold)
    {
        assert(src.data);
        assert(src.width);
        assert(src.height);
        assert(dst.data);
        assert(dst.width == src.width);
        assert(dst.height == src.height);

        u32 n_elements = src.width * src.height;
        int threads_per_block = THREADS_PER_BLOCK;
        int blocks = (n_elements + threads_per_block - 1) / threads_per_block;

        bool proc;

        proc = cuda_no_errors();
        assert(proc);

        proc &= cuda_no_errors();
        assert(proc);

        gpu_edges<<<blocks, threads_per_block>>>(
            src.data,
            dst.data,
            src.width,
            src.height,
            threshold);

        proc &= cuda_launch_success();
        assert(proc);

        return proc;
    }


    bool gradients(gray::device_image_t const& src, gray::device_image_t const& dst, gray::device_image_t const& temp)
    {
        assert(src.data);
        assert(src.width);
        assert(src.height);
        assert(dst.data);
        assert(dst.width == src.width);
        assert(dst.height == src.height);
        assert(temp.data);
        assert(temp.width == src.width);
        assert(temp.height == src.height);

        u32 n_elements = src.width * src.height;
        int threads_per_block = THREADS_PER_BLOCK;
        int blocks = (n_elements + threads_per_block - 1) / threads_per_block;

        bool proc;

        proc = cuda_no_errors();
        assert(proc);

        gpu_blur<<<blocks, threads_per_block>>>(
            src.data, 
            temp.data,
            src.width, 
            src.height);
        
        proc &= cuda_launch_success();
        assert(proc);

        proc &= cuda_no_errors();
        assert(proc);

        gpu_gradients<<<blocks, threads_per_block>>>(
            temp.data,
            dst.data,
            src.width,
            src.height);

        proc &= cuda_launch_success();
        assert(proc);

        return proc;
    }


    bool gradients(gray::device_image_t const& src, gray::device_image_t const& dst)
    {
        assert(src.data);
        assert(src.width);
        assert(src.height);
        assert(dst.data);
        assert(dst.width == src.width);
        assert(dst.height == src.height);

        u32 n_elements = src.width * src.height;
        int threads_per_block = THREADS_PER_BLOCK;
        int blocks = (n_elements + threads_per_block - 1) / threads_per_block;

        bool proc;

        proc = cuda_no_errors();
        assert(proc);

        proc &= cuda_no_errors();
        assert(proc);

        gpu_gradients<<<blocks, threads_per_block>>>(
            src.data,
            dst.data,
            src.width,
            src.height);

        proc &= cuda_launch_success();
        assert(proc);

        return proc;
    }


    bool contrast(gray::device_image_t const& src, gray::device_image_t const& dst, u8 src_low, u8 src_high)
    {
        assert(src.data);
        assert(src.width);
        assert(src.height);
        assert(dst.data);
        assert(dst.width == src.width);
        assert(dst.height == src.height);

        u32 n_elements = src.width * src.height;
        int threads_per_block = THREADS_PER_BLOCK;
        int blocks = (n_elements + threads_per_block - 1) / threads_per_block;

        bool proc;

        proc = cuda_no_errors();
        assert(proc);

        gpu_transform_contrast<<<blocks, threads_per_block>>>(
            src.data,
            dst.data,
            src_low,
            src_high,
            n_elements);

        proc &= cuda_launch_success();
        assert(proc);

        return proc;
    }
        

#endif // !LIBIMAGE_NO_GRAYSCALE


#ifndef LIBIMAGE_NO_COLOR
#ifndef LIBIMAGE_NO_GRAYSCALE
        

    bool grayscale(device_image_t const& src, gray::device_image_t const& dst)
    {
        assert(src.data);
        assert(src.width);
        assert(src.height);
        assert(dst.data);
        assert(dst.width == src.width);
        assert(dst.height == src.height);

        u32 n_elements = src.width * src.height;
        int threads_per_block = THREADS_PER_BLOCK;
        int blocks = (n_elements + threads_per_block - 1) / threads_per_block;

        bool proc;

        proc = cuda_no_errors();
        assert(proc);

        gpu_transform_grayscale<<<blocks, threads_per_block>>>(
            src.data, 
            dst.data, 
            n_elements);

        proc &= cuda_launch_success();
        assert(proc);

        return proc;
    }

#endif // !LIBIMAGE_NO_GRAYSCALE
#endif // !LIBIMAGE_NO_COLOR	
    
}