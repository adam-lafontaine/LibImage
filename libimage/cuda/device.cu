/*

Copyright (c) 2021 Adam Lafontaine

*/
#include "device.hpp"
#include "cuda_def.cuh"

#include <iostream>


static void check_error(hipError_t err)
{
    if(err == hipSuccess)
    {
        return;
    }

    std::cout 
        << "\n*** CUDA ERROR ***\n\n" 
        << hipGetErrorString(err)
        << "\n\n******************\n\n";
}


bool cuda_device_malloc(void** ptr, u32 n_bytes)
{
    hipError_t err = hipMalloc(ptr, n_bytes);
    check_error(err);
    
    return err == hipSuccess;
}


bool cuda_device_free(void* ptr)
{
    hipError_t err = hipFree(ptr);
    check_error(err);

    return err == hipSuccess;
}


bool cuda_memcpy_to_device(const void* host_src, void* device_dst, size_t n_bytes)
{
    hipError_t err = hipMemcpy(device_dst, host_src, n_bytes, hipMemcpyHostToDevice);
    check_error(err);

    return err == hipSuccess;
}


bool cuda_memcpy_to_host(const void* device_src, void* host_dst, size_t n_bytes)
{
    hipError_t err = hipMemcpy(host_dst, device_src, n_bytes, hipMemcpyDeviceToHost);
    check_error(err);

    return err == hipSuccess;
}


bool cuda_no_errors()
{
    hipError_t err = hipGetLastError();
    check_error(err);

    return err == hipSuccess;
}


bool cuda_launch_success()
{
    hipError_t err = hipDeviceSynchronize();
    check_error(err);

    return err == hipSuccess;
}