#include "device.hpp"
#include "cuda_def.cuh"

#ifdef CUDA_PRINT_ERROR

#include <cstdio>
#include <cassert>

#endif


static void check_error(hipError_t err)
{
    if(err == hipSuccess)
    {
        return;
    }

    #ifdef CUDA_PRINT_ERROR

    printf("\n*** CUDA ERROR ***\n\n");
    printf("%s", hipGetErrorString(err));
    printf("\n\n******************\n\n");

    #endif
}


bool cuda_device_malloc(void** ptr, u32 n_bytes)
{
    hipError_t err = hipMalloc(ptr, n_bytes);
    check_error(err);
    
    return err == hipSuccess;
}


bool cuda_device_free(void* ptr)
{
    hipError_t err = hipFree(ptr);
    check_error(err);

    return err == hipSuccess;
}


bool cuda_memcpy_to_device(const void* host_src, void* device_dst, size_t n_bytes)
{
    hipError_t err = hipMemcpy(device_dst, host_src, n_bytes, hipMemcpyHostToDevice);
    check_error(err);

    return err == hipSuccess;
}


bool cuda_memcpy_to_host(const void* device_src, void* host_dst, size_t n_bytes)
{
    hipError_t err = hipMemcpy(host_dst, device_src, n_bytes, hipMemcpyDeviceToHost);
    check_error(err);

    return err == hipSuccess;
}


bool cuda_no_errors()
{
    hipError_t err = hipGetLastError();
    check_error(err);

    return err == hipSuccess;
}


bool cuda_launch_success()
{
    hipError_t err = hipDeviceSynchronize();
    check_error(err);

    return err == hipSuccess;
}


namespace device
{
    bool malloc(MemoryBuffer& buffer, size_t n_bytes)
    {
        assert(!buffer.data);

        hipError_t err = hipMalloc((void**)&(buffer.data), n_bytes);
        check_error(err);

        bool result = err == hipSuccess;

        if(result)
        {
            buffer.capacity = n_bytes;
        }
        
        return result;
    }


    bool free(MemoryBuffer& buffer)
    {
        buffer.capacity = 0;
        buffer.size = 0;

        if(buffer.data)
        {
            hipError_t err = hipFree(buffer.data);
            check_error(err);

            return err == hipSuccess;
        }

        return true;
    }


    u8* push_bytes(MemoryBuffer& buffer, size_t n_bytes)
    {
        assert(buffer.data);
        assert(buffer.capacity);
        assert(buffer.size < buffer.capacity);

        auto is_valid = 
            buffer.data &&
            buffer.capacity &&
            buffer.size < buffer.capacity;

        auto bytes_available = (buffer.capacity - buffer.size) >= n_bytes;
        assert(bytes_available);

        if(!is_valid || !bytes_available)
        {
            return nullptr;
        }

        auto data = buffer.data + buffer.size;

        buffer.size += n_bytes;

        return data;
    }


    bool pop_bytes(MemoryBuffer& buffer, size_t n_bytes)
    {
        assert(buffer.data);
        assert(buffer.capacity);
        assert(buffer.size <= buffer.capacity);
        assert(n_bytes <= buffer.capacity);
        assert(n_bytes <= buffer.size);

        auto is_valid = 
            buffer.data &&
            buffer.capacity &&
            buffer.size <= buffer.capacity &&
            n_bytes <= buffer.capacity &&
            n_bytes <= buffer.size;

        if(is_valid)
        {
            buffer.size -= n_bytes;
            return true;
        }

        return false;
    }
}