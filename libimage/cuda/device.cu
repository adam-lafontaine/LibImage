#include "device.hpp"
#include "cuda_def.cuh"

#ifdef CUDA_PRINT_ERROR

#include <cstdio>
#include <cassert>

#endif


static void check_error(hipError_t err)
{
    if(err == hipSuccess)
    {
        return;
    }

    #ifdef CUDA_PRINT_ERROR

    printf("\n*** CUDA ERROR ***\n\n");
    printf("%s", hipGetErrorString(err));
    printf("\n\n******************\n\n");

    #endif
}


bool cuda_device_malloc(void** ptr, u32 n_bytes)
{
    hipError_t err = hipMalloc(ptr, n_bytes);
    check_error(err);
    
    return err == hipSuccess;
}


bool cuda_device_free(void* ptr)
{
    hipError_t err = hipFree(ptr);
    check_error(err);

    return err == hipSuccess;
}


bool cuda_memcpy_to_device(const void* host_src, void* device_dst, size_t n_bytes)
{
    hipError_t err = hipMemcpy(device_dst, host_src, n_bytes, hipMemcpyHostToDevice);
    check_error(err);

    return err == hipSuccess;
}


bool cuda_memcpy_to_host(const void* device_src, void* host_dst, size_t n_bytes)
{
    hipError_t err = hipMemcpy(host_dst, device_src, n_bytes, hipMemcpyDeviceToHost);
    check_error(err);

    return err == hipSuccess;
}


bool cuda_no_errors()
{
    hipError_t err = hipGetLastError();
    check_error(err);

    return err == hipSuccess;
}


bool cuda_launch_success()
{
    hipError_t err = hipDeviceSynchronize();
    check_error(err);

    return err == hipSuccess;
}


namespace device
{
    bool malloc(MemoryBuffer& buffer, size_t n_bytes)
    {
        assert(!buffer.data);

        buffer.size = 0;

        hipError_t err = hipMalloc((void**)&(buffer.data), n_bytes);
        check_error(err);

        bool result = err == hipSuccess;
        //bool result = cuda_device_malloc((void**)&(buffer.data), n_bytes);

        if(result)
        {
            buffer.capacity = n_bytes;
        }
        
        return result;
    }


    bool free(MemoryBuffer& buffer)
    {
        buffer.capacity = 0;
        buffer.size = 0;

        if(buffer.data)
        {
            hipError_t err = hipFree(buffer.data);
            check_error(err);

            return err == hipSuccess;
        }

        return true;
    }


    u8* push(MemoryBuffer& buffer, size_t n_bytes)
    {
        assert(is_valid(buffer));

        auto bytes_available = buffer.capacity - buffer.size;
        assert(bytes_available >= n_bytes);

        if(!is_valid(buffer) || n_bytes > bytes_available)
        {
            return nullptr;
        }

        auto data = buffer.data + buffer.size;

        buffer.size += n_bytes;

        return data;
    }


    bool pop(MemoryBuffer& buffer, size_t n_bytes)
    {
        assert(buffer.data);
        assert(buffer.capacity);
        assert(buffer.size <= buffer.capacity);
        assert(n_bytes <= buffer.capacity);
        assert(n_bytes <= buffer.size);

        auto is_valid = 
            buffer.data &&
            buffer.capacity &&
            buffer.size <= buffer.capacity &&
            n_bytes <= buffer.capacity &&
            n_bytes <= buffer.size;

        if(is_valid)
        {
            buffer.size -= n_bytes;
            return true;
        }

        return false;
    }


    bool is_valid(MemoryBuffer const& buffer)
    {
        return 
            buffer.data &&
            buffer.capacity &&
            buffer.size < buffer.capacity;
    }
}