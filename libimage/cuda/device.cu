#include "device.hpp"
#include "cuda_def.cuh"


static bool device_malloc(void** ptr, u32 n_bytes)
{
    hipError_t err = hipMalloc(ptr, n_bytes);
    return err == hipSuccess;
}


static bool device_free(void* ptr)
{
    hipError_t err = hipFree(ptr);
    return err == hipSuccess;
}


bool device_malloc(DeviceBuffer& buffer, size_t n_bytes)
{
    bool result = device_malloc((void**)&(buffer.data), n_bytes);
    if(result)
    {
        buffer.total_bytes = n_bytes;
    }

    return result;
}


bool device_free(DeviceBuffer& buffer)
{
    buffer.total_bytes = 0;
    buffer.offset = 0;
    return device_free(buffer.data);
}


bool memcpy_to_device(const void* host_src, void* device_dst, size_t n_bytes)
{
    hipError_t err = hipMemcpy(device_dst, host_src, n_bytes, hipMemcpyHostToDevice);
    return err == hipSuccess;
}


bool memcpy_to_host(const void* device_src, void* host_dst, size_t n_bytes)
{
    hipError_t err = hipMemcpy(host_dst, device_src, n_bytes, hipMemcpyDeviceToHost);
    return err == hipSuccess;
}