#include "device.hpp"
#include "cuda_def.cuh"

#ifdef CUDA_PRINT_ERROR

#include <cstdio>
#include <cassert>

#endif


static void check_error(hipError_t err)
{
    if(err == hipSuccess)
    {
        return;
    }

    #ifdef CUDA_PRINT_ERROR

    printf("\n*** CUDA ERROR ***\n\n");
    printf("%s", hipGetErrorString(err));
    printf("\n\n******************\n\n");

    #endif
}


bool cuda_device_malloc(void** ptr, u32 n_bytes)
{
    hipError_t err = hipMalloc(ptr, n_bytes);
    check_error(err);
    
    return err == hipSuccess;
}


bool cuda_device_free(void* ptr)
{
    hipError_t err = hipFree(ptr);
    check_error(err);

    return err == hipSuccess;
}


bool cuda_memcpy_to_device(const void* host_src, void* device_dst, size_t n_bytes)
{
    hipError_t err = hipMemcpy(device_dst, host_src, n_bytes, hipMemcpyHostToDevice);
    check_error(err);

    return err == hipSuccess;
}


bool cuda_memcpy_to_host(const void* device_src, void* host_dst, size_t n_bytes)
{
    hipError_t err = hipMemcpy(host_dst, device_src, n_bytes, hipMemcpyDeviceToHost);
    check_error(err);

    return err == hipSuccess;
}


bool cuda_no_errors()
{
    hipError_t err = hipGetLastError();
    check_error(err);

    return err == hipSuccess;
}


bool cuda_launch_success()
{
    hipError_t err = hipDeviceSynchronize();
    check_error(err);

    return err == hipSuccess;
}


bool device_malloc(DeviceBuffer& buffer, size_t n_bytes)
{
    bool result = cuda_device_malloc((void**)&(buffer.data), n_bytes);
    if(result)
    {
        buffer.total_bytes = n_bytes;
    }

    return result;
}


bool device_free(DeviceBuffer& buffer)
{
    buffer.total_bytes = 0;
    buffer.offset = 0;
    return cuda_device_free(buffer.data);
}


namespace device
{
    bool malloc(MemoryBuffer& buffer, size_t n_bytes)
    {
        assert(!buffer.data);

        buffer.offset = 0;

        hipError_t err = hipMalloc((void**)&(buffer.data), n_bytes);
        check_error(err);

        bool result = err == hipSuccess;

        if(result)
        {
            buffer.capacity = n_bytes;
        }
        
        return result;
    }


    bool free(MemoryBuffer& buffer)
    {
        buffer.capacity = 0;
        buffer.offset = 0;

        hipError_t err = hipFree(buffer.data);
        check_error(err);

        return err == hipSuccess;
    }


    u8* push(MemoryBuffer& buffer, size_t n_bytes)
    {
        assert(is_valid(buffer));

        auto bytes_available = buffer.capacity - buffer.offset;
        assert(bytes_available >= n_bytes);

        if(!is_valid(buffer) || n_bytes > bytes_available)
        {
            return nullptr;
        }

        buffer.offset += n_bytes;

        return buffer.data + buffer.offset;
    }


    bool is_valid(MemoryBuffer const& buffer)
    {
        return 
            buffer.data &&
            buffer.capacity &&
            buffer.offset < buffer.capacity;
    }
}